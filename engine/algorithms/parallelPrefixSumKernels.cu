#include "hip/hip_runtime.h"
#include "parallelPrefixSumKernels.hu"

__global__ void parallelPrefix(uint numElements, uint* array, uint* blockSums){ // BLOCKSIZE threads operating on WORKSIZE (2x BLOCKSIZE) elements
    uint index = threadIdx.x + blockIdx.x*WORKSIZE;
    __shared__ uint shared[WORKSIZE];
    __shared__ uint blockSum;
    if(index < numElements){
        shared[threadIdx.x] = array[index];
    }
    else{
        shared[threadIdx.x] = 0;
    }
    if(index + blockDim.x < numElements){
        shared[threadIdx.x + blockDim.x] = array[index + blockDim.x];
    }
    else{
        shared[threadIdx.x + blockDim.x] = 0;
    }
    blockWiseExclusivePrefixSum(shared, WORKSIZE, blockSum);
    blockSums[blockIdx.x] = blockSum;
    if(index < numElements){    //copy first of the two values handled by each thread
        array[index] = shared[threadIdx.x + 1];
    }
    if(index + blockDim.x < numElements){
        if(threadIdx.x == blockDim.x-1){
            array[index + blockDim.x] = blockSum;
        }
        else {
            array[index + blockDim.x] = shared[threadIdx.x + blockDim.x + 1];
        }

    }
}

__global__ void parallelPrefixApplyPreviousBlockSum(uint numElements, uint* array, uint* blockSums){ //designed for WORKSIZE threads per block, 2x BLOCKSIZE
    uint index = threadIdx.x + (blockIdx.x)*blockDim.x;
    if(blockIdx.x > 0 && index < numElements){    //STORE
        array[index] += blockSums[blockIdx.x - 1];
    }
}