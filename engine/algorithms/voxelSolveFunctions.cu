#include "hip/hip_runtime.h"
//Copyright 2023 Aberrant Behavior LLC

#include "voxelSolveFunctions.hu"

__device__ float weightFromDistance(float in, float radius){
    if(in > radius){
        return 0.0f;
    }
    else{
        return 1.0f - in/radius;
    }
}

__global__ void voxelUGather(uint numVoxelsPerNode, uint numUsedVoxelsInGrid, uint numParticlesInParticleLists, uint* gridPosition, float* particleVs,
    float* px, float* py, float* pz, uint* nodeIndexToFirstVoxelIndex, uint* voxelIDs, uint* perVoxelParticleListStartIndices, uint* particleLists,
    float* voxelUs, Grid grid, uint xySize, uint refinementLevel, float radius, VelocityGatherDimension solveDimension, uint numVoxels1D)
{
    extern __shared__ float thisNodeU[];
    __shared__ uint maxVoxelIndex;
    __shared__ uint voxelStartIndex;
    __shared__ uint thisNodeParticleListStopIndex;
    __shared__ uint thisNodeParticleListStartIndex;

    for(uint i = threadIdx.x; i < numVoxelsPerNode; i += blockDim.x){
        thisNodeU[i] = 0.0f;
    }
    if(threadIdx.x == 0){
        maxVoxelIndex = nodeIndexToFirstVoxelIndex[blockIdx.x];
        if(maxVoxelIndex < numUsedVoxelsInGrid - 1){
            thisNodeParticleListStopIndex = perVoxelParticleListStartIndices[maxVoxelIndex];
        }
        else{
            thisNodeParticleListStopIndex = numParticlesInParticleLists;
        }
        if(blockIdx.x == 0){
            voxelStartIndex = 0;
        }
        else{
            voxelStartIndex = nodeIndexToFirstVoxelIndex[blockIdx.x - 1];
        }
        thisNodeParticleListStartIndex = perVoxelParticleListStartIndices[voxelStartIndex];
    }
    __syncthreads();
    float subCellWidth = grid.cellSize/(2.0f*(1<<refinementLevel));
    uint apronCells = floorf(radius);
    uint numVoxels2D = numVoxels1D*numVoxels1D;
    for(uint curParticleInNode = thisNodeParticleListStartIndex + threadIdx.x; curParticleInNode < thisNodeParticleListStopIndex; curParticleInNode += blockDim.x){
        uint curThreadParticleVoxelID;
        // uint curThreadParticleVoxelListIndex;
        uint particleIndex = particleLists[curParticleInNode];
        for(uint curVoxelIndex = voxelStartIndex; curVoxelIndex < maxVoxelIndex; ++curVoxelIndex){
            uint voxelParticleListStart = perVoxelParticleListStartIndices[curVoxelIndex];
            uint voxelParticleListStop;
            if(curVoxelIndex < numUsedVoxelsInGrid - 1){
                voxelParticleListStop = perVoxelParticleListStartIndices[curVoxelIndex+1];
            }
            else{
                voxelParticleListStop = numParticlesInParticleLists;
            }
            if(curParticleInNode >= voxelParticleListStart && curParticleInNode < voxelParticleListStop){
                curThreadParticleVoxelID = voxelIDs[curVoxelIndex];
            }
        }
        uint voxelIDx = curThreadParticleVoxelID % numVoxels1D;
        uint voxelIDy = (curThreadParticleVoxelID % numVoxels2D) / numVoxels1D;
        uint voxelIDz = curThreadParticleVoxelID / numVoxels2D;

        float voxelPx = voxelIDx * subCellWidth;
        float voxelPy = voxelIDy * subCellWidth;
        float voxelPz = voxelIDz * subCellWidth;
        
        if(solveDimension == VelocityGatherDimension::X){
            voxelPy += 0.5*subCellWidth;
            voxelPz += 0.5*subCellWidth;
        }
        else if(solveDimension == VelocityGatherDimension::Y){
            voxelPx += 0.5*subCellWidth;
            voxelPz += 0.5*subCellWidth;
        }
        else{
            voxelPx += 0.5*subCellWidth;
            voxelPy += 0.5*subCellWidth;
        }

        uint moduloWRTxySize = gridPosition[particleIndex] % xySize;
        uint gridIDz = gridPosition[particleIndex] / (xySize);
        uint gridIDy = (moduloWRTxySize) / grid.sizeX;
        uint gridIDx = (moduloWRTxySize) % grid.sizeX;
        
        float pxInGridCell = (px[particleIndex] - grid.negX - gridIDx*grid.cellSize + apronCells*subCellWidth);
        float pyInGridCell = (py[particleIndex] - grid.negY - gridIDy*grid.cellSize + apronCells*subCellWidth);
        float pzInGridCell = (pz[particleIndex] - grid.negZ - gridIDz*grid.cellSize + apronCells*subCellWidth);

        float dpx = pxInGridCell - voxelPx;
        float dpy = pyInGridCell - voxelPy;
        float dpz = pzInGridCell - voxelPz;
        float dpx_2 = dpx*dpx;
        float dpy_2 = dpy*dpy;
        float dpz_2 = dpz*dpz;

        float particleToNodeDistance = sqrtf(dpx_2 + dpy_2 + dpz_2) / subCellWidth;

        atomicAdd(thisNodeU + curThreadParticleVoxelID, weightFromDistance(particleToNodeDistance, radius) * particleVs[particleIndex]);
    }
    __syncthreads();
    for(uint usedVoxelIndex = voxelStartIndex + threadIdx.x; usedVoxelIndex < maxVoxelIndex; usedVoxelIndex += blockDim.x){
        voxelUs[usedVoxelIndex] = thisNodeU[voxelIDs[usedVoxelIndex]];
    }
}

void cudaVoxelUGather(uint numUsedVoxelsGrid, uint numGridNodes, uint numParticles, uint numVoxelsPerNode, uint numParticlesInParticleLists,
    uint* gridPosition, float* particleVs, float* px, float* py, float* pz, uint* nodeIndexToFirstVoxelIndex, uint* voxelIDs,
    uint* perVoxelParticleListStartIndices, uint* particleLists, float* voxelUs, Grid grid, uint xySize, uint refinementLevel, float radius,
    uint numVoxels1D, VelocityGatherDimension solveDimension, hipStream_t stream)
{
    voxelUGather<<<numGridNodes, BLOCKSIZE, sizeof(float) * numVoxelsPerNode, stream>>>(numVoxelsPerNode, numUsedVoxelsGrid, numParticlesInParticleLists,
        gridPosition, particleVs, px, py, pz, nodeIndexToFirstVoxelIndex, voxelIDs, perVoxelParticleListStartIndices, particleLists, voxelUs, grid, xySize,
        refinementLevel, radius, solveDimension, numVoxels1D);
}

__device__ bool isIndexApronCell(uint voxelIndex, const uint& numVoxels1D, const uint& refinementLevel, const float& radius){
    uint numApronCellsAtBorder = floorf(radius);
    uint rightBoundApronCells = (1<<refinementLevel) + numApronCellsAtBorder;
    uint xWiseIndex = voxelIndex % (numVoxels1D);
    uint yWiseIndex = (voxelIndex / numVoxels1D) % numVoxels1D;
    uint zWiseIndex = voxelIndex / (numVoxels1D*numVoxels1D);
    return xWiseIndex < numApronCellsAtBorder
        || xWiseIndex > rightBoundApronCells
        || yWiseIndex < numApronCellsAtBorder
        || yWiseIndex > rightBoundApronCells
        || zWiseIndex < numApronCellsAtBorder
        || zWiseIndex > rightBoundApronCells;
}

__device__ void addVoxelDataForUsedNode(uint thisThreadNodeIndexToHandle, float* sharedBlockVoxelStorage, uint* nodeIndexToFirstVoxelIndex, uint* voxelIDs, float* voxelData){
    uint startVoxelID;
    if(thisThreadNodeIndexToHandle == 0){
        startVoxelID = 0;
    }
    else{
        startVoxelID = nodeIndexToFirstVoxelIndex[thisThreadNodeIndexToHandle-1];
    }
    for(uint currentUsedVoxelIndex = startVoxelID + threadIdx.x; currentUsedVoxelIndex < nodeIndexToFirstVoxelIndex[thisThreadNodeIndexToHandle]; currentUsedVoxelIndex += blockDim.x){
        sharedBlockVoxelStorage[voxelIDs[currentUsedVoxelIndex]] += voxelData[voxelIDs[currentUsedVoxelIndex]];
    }

}

__device__ void loadVoxelDataForThisBlock(uint numVoxelsPerNode, uint numUniqueGridNodes, float* sharedBlockStorage, uint* nodeIndexToFirstVoxelIndex, uint* voxelIDs,
    float* voxelData, float radius, uint refinementLevel, Grid grid, uint* yDimToFirstNodeIndex, uint* gridNodeIndicesToFirstParticleIndex, uint* gridNodeIDs)
{
    uint index = threadIdx.x + blockIdx.x*blockDim.x;
    if(index < numUniqueGridNodes){
        for(uint i = threadIdx.x; i < numVoxelsPerNode; i += blockDim.x){
            sharedBlockStorage[i] = 0.0f;
        }
        __syncthreads();
        addVoxelDataForUsedNode(index, sharedBlockStorage, nodeIndexToFirstVoxelIndex, voxelIDs, voxelData);
        uint curGridNodeID = gridNodeIDs[gridNodeIndicesToFirstParticleIndex[blockIdx.x]];
        uint curY = (curGridNodeID / grid.sizeX) % grid.sizeY;
        uint curZ = curGridNodeID / (grid.sizeX*grid.sizeY);

        uint yNeighborLeft = curY - 1;
        uint yNeighborRight = curY + 1;
        uint zNeighborLeft = curZ - 1;
        uint zNeighborRight = curZ + 1;

        __syncthreads();
        uint neighborUniqueID;
        if((neighborUniqueID = yDimToFirstNodeIndex[yNeighborLeft + curZ*grid.sizeY]) < numUniqueGridNodes){
            addVoxelDataForUsedNode(neighborUniqueID, sharedBlockStorage, nodeIndexToFirstVoxelIndex, voxelIDs, voxelData);
        }
        __syncthreads();
        if((neighborUniqueID = yDimToFirstNodeIndex[yNeighborRight + curZ*grid.sizeY]) < numUniqueGridNodes){
            addVoxelDataForUsedNode(neighborUniqueID, sharedBlockStorage, nodeIndexToFirstVoxelIndex, voxelIDs, voxelData);
        }
        __syncthreads();
        if((neighborUniqueID = yDimToFirstNodeIndex[zNeighborRight*grid.sizeY + curY]) < numUniqueGridNodes){
            addVoxelDataForUsedNode(neighborUniqueID, sharedBlockStorage, nodeIndexToFirstVoxelIndex, voxelIDs, voxelData);
        }
        __syncthreads();
        if((neighborUniqueID = yDimToFirstNodeIndex[zNeighborLeft*grid.sizeY + curY]) < numUniqueGridNodes){
            addVoxelDataForUsedNode(neighborUniqueID, sharedBlockStorage, nodeIndexToFirstVoxelIndex, voxelIDs, voxelData);
        }
        __syncthreads();
    }
}

__global__ void calculateDivU(uint numVoxelsPerNode, uint numUsedVoxelsGrid, uint* nodeIndexToFirstVoxelIndex, uint* voxelIDs, float* voxelUs,
    Grid grid, uint* yDimFirstNodeIndex)
{
    extern __shared__ float sharedVoxels[];


}