#include "hip/hip_runtime.h"
//Copyright 2023 Aberrant Behavior LLC

#include "voxelSolveFunctions.hu"

__device__ float weightFromDistance(float in, float radius){
    if(in > radius){
        return 0.0f;
    }
    else{
        return 1.0f - in/radius;
    }
}

__global__ void voxelUGather(uint numVoxelsPerNode, uint numUsedVoxelsInGrid, uint numParticlesInParticleLists, uint* gridPosition, float* particleVs,
    float* px, float* py, float* pz, uint* nodeIndexToFirstVoxelIndex, uint* voxelIDs, uint* perVoxelParticleListStartIndices, uint* particleLists,
    float* voxelUs, Grid grid, uint xySize, uint refinementLevel, float radius, VelocityGatherDimension solveDimension, uint numVoxels1D)
{
    extern __shared__ float thisNodeU[];
    __shared__ uint maxVoxelIndex;
    __shared__ uint voxelStartIndex;
    __shared__ uint thisNodeParticleListStopIndex;
    __shared__ uint thisNodeParticleListStartIndex;

    for(uint i = threadIdx.x; i < numVoxelsPerNode; i += blockDim.x){
        thisNodeU[i] = 0.0f;
    }
    if(threadIdx.x == 0){
        maxVoxelIndex = nodeIndexToFirstVoxelIndex[blockIdx.x];
        if(maxVoxelIndex < numUsedVoxelsInGrid - 1){
            thisNodeParticleListStopIndex = perVoxelParticleListStartIndices[maxVoxelIndex];
        }
        else{
            thisNodeParticleListStopIndex = numParticlesInParticleLists;
        }
        if(blockIdx.x == 0){
            voxelStartIndex = 0;
        }
        else{
            voxelStartIndex = nodeIndexToFirstVoxelIndex[blockIdx.x - 1];
        }
        thisNodeParticleListStartIndex = perVoxelParticleListStartIndices[voxelStartIndex];
    }
    __syncthreads();
    float subCellWidth = grid.cellSize/(2.0f*(1<<refinementLevel));
    uint apronCells = floorf(radius);
    uint numVoxels2D = numVoxels1D*numVoxels1D;
    for(uint curParticleInNode = thisNodeParticleListStartIndex + threadIdx.x; curParticleInNode < thisNodeParticleListStopIndex; curParticleInNode += blockDim.x){
        uint curThreadParticleVoxelID;
        // uint curThreadParticleVoxelListIndex;
        uint particleIndex = particleLists[curParticleInNode];
        for(uint curVoxelIndex = voxelStartIndex; curVoxelIndex < maxVoxelIndex; ++curVoxelIndex){
            uint voxelParticleListStart = perVoxelParticleListStartIndices[curVoxelIndex];
            uint voxelParticleListStop;
            if(curVoxelIndex < numUsedVoxelsInGrid - 1){
                voxelParticleListStop = perVoxelParticleListStartIndices[curVoxelIndex+1];
            }
            else{
                voxelParticleListStop = numParticlesInParticleLists;
            }
            if(curParticleInNode >= voxelParticleListStart && curParticleInNode < voxelParticleListStop){
                curThreadParticleVoxelID = voxelIDs[curVoxelIndex];
            }
        }
        uint voxelIDx = curThreadParticleVoxelID % numVoxels1D;
        uint voxelIDy = (curThreadParticleVoxelID % numVoxels2D) / numVoxels1D;
        uint voxelIDz = curThreadParticleVoxelID / numVoxels2D;

        float voxelPx = voxelIDx * subCellWidth;
        float voxelPy = voxelIDy * subCellWidth;
        float voxelPz = voxelIDz * subCellWidth;
        
        if(solveDimension == VelocityGatherDimension::X){
            voxelPy += 0.5*subCellWidth;
            voxelPz += 0.5*subCellWidth;
        }
        else if(solveDimension == VelocityGatherDimension::Y){
            voxelPx += 0.5*subCellWidth;
            voxelPz += 0.5*subCellWidth;
        }
        else{
            voxelPx += 0.5*subCellWidth;
            voxelPy += 0.5*subCellWidth;
        }

        uint moduloWRTxySize = gridPosition[particleIndex] % xySize;
        uint gridIDz = gridPosition[particleIndex] / (xySize);
        uint gridIDy = (moduloWRTxySize) / grid.sizeX;
        uint gridIDx = (moduloWRTxySize) % grid.sizeX;
        
        float pxInGridCell = (px[particleIndex] - grid.negX - gridIDx*grid.cellSize + apronCells*subCellWidth);
        float pyInGridCell = (py[particleIndex] - grid.negY - gridIDy*grid.cellSize + apronCells*subCellWidth);
        float pzInGridCell = (pz[particleIndex] - grid.negZ - gridIDz*grid.cellSize + apronCells*subCellWidth);

        float dpx = pxInGridCell - voxelPx;
        float dpy = pyInGridCell - voxelPy;
        float dpz = pzInGridCell - voxelPz;
        float dpx_2 = dpx*dpx;
        float dpy_2 = dpy*dpy;
        float dpz_2 = dpz*dpz;

        float particleToNodeDistance = sqrtf(dpx_2 + dpy_2 + dpz_2) / subCellWidth;

        atomicAdd(thisNodeU + curThreadParticleVoxelID, weightFromDistance(particleToNodeDistance, radius) * particleVs[particleIndex]);
    }
    __syncthreads();
    for(uint usedVoxelIndex = voxelStartIndex + threadIdx.x; usedVoxelIndex < maxVoxelIndex; usedVoxelIndex += blockDim.x){
        voxelUs[usedVoxelIndex] = thisNodeU[voxelIDs[usedVoxelIndex]];
    }
}

void cudaVoxelUGather(uint numUsedVoxelsGrid, uint numGridNodes, uint numParticles, uint numVoxelsPerNode, uint numParticlesInParticleLists,
    uint* gridPosition, float* particleVs, float* px, float* py, float* pz, uint* nodeIndexToFirstVoxelIndex, uint* voxelIDs,
    uint* perVoxelParticleListStartIndices, uint* particleLists, float* voxelUs, Grid grid, uint xySize, uint refinementLevel, float radius,
    uint numVoxels1D, VelocityGatherDimension solveDimension, hipStream_t stream)
{
    voxelUGather<<<numGridNodes, BLOCKSIZE, sizeof(float) * numVoxelsPerNode, stream>>>(numVoxelsPerNode, numUsedVoxelsGrid, numParticlesInParticleLists,
        gridPosition, particleVs, px, py, pz, nodeIndexToFirstVoxelIndex, voxelIDs, perVoxelParticleListStartIndices, particleLists, voxelUs, grid, xySize,
        refinementLevel, radius, solveDimension, numVoxels1D);
}

__device__ bool isIndexApronCell(uint voxelIndex, const uint& numVoxels1D, const uint& refinementLevel, const float& radius){
    uint numApronCellsAtBorder = floorf(radius);
    uint rightBoundApronCells = (1<<refinementLevel) + numApronCellsAtBorder;
    uint xWiseIndex = voxelIndex % (numVoxels1D);
    uint yWiseIndex = (voxelIndex / numVoxels1D) % numVoxels1D;
    uint zWiseIndex = voxelIndex / (numVoxels1D*numVoxels1D);
    return xWiseIndex < numApronCellsAtBorder
        || xWiseIndex > rightBoundApronCells
        || yWiseIndex < numApronCellsAtBorder
        || yWiseIndex > rightBoundApronCells
        || zWiseIndex < numApronCellsAtBorder
        || zWiseIndex > rightBoundApronCells;
}

__device__ void addVoxelDataForUsedNode(const uint thisThreadNodeIndexToHandle, float* sharedBlockVoxelStorage, const uint* nodeIndexToFirstVoxelIndex,
        const uint* voxelIDs, const float* voxelData)
{
    uint startVoxelID;
    if(thisThreadNodeIndexToHandle == 0){
        startVoxelID = 0;
    }
    else{
        startVoxelID = nodeIndexToFirstVoxelIndex[thisThreadNodeIndexToHandle-1];
    }
    for(uint currentUsedVoxelIndex = startVoxelID + threadIdx.x; currentUsedVoxelIndex < nodeIndexToFirstVoxelIndex[thisThreadNodeIndexToHandle]; currentUsedVoxelIndex += blockDim.x){
        sharedBlockVoxelStorage[voxelIDs[currentUsedVoxelIndex]] += voxelData[voxelIDs[currentUsedVoxelIndex]];
    }

}

__device__ void loadVoxelDataForThisBlock(const uint numVoxelsPerNode, const uint numUniqueGridNodes, float* sharedBlockStorage,
    const uint* nodeIndexToFirstVoxelIndex, const uint* voxelIDs, const float* voxelData, const float radius, const uint refinementLevel,
    const Grid& grid, const uint* yDimToFirstNodeIndex, const uint* gridNodeIndicesToFirstParticleIndex, const uint* gridNodeIDs)
{
    for(uint i = threadIdx.x; i < numVoxelsPerNode; i += blockDim.x){
        sharedBlockStorage[i] = 0.0f;
    }
    __syncthreads();
    addVoxelDataForUsedNode(threadIdx.x + blockIdx.x*blockDim.x, sharedBlockStorage,
        nodeIndexToFirstVoxelIndex, voxelIDs, voxelData);
    
    uint curGridNodeID = gridNodeIDs[gridNodeIndicesToFirstParticleIndex[blockIdx.x]];
    uint curY = (curGridNodeID / grid.sizeX) % grid.sizeY;
    uint curZ = curGridNodeID / (grid.sizeX*grid.sizeY);

    uint yNeighborLeft = curY - 1;
    uint yNeighborRight = curY + 1;
    uint zNeighborLeft = curZ - 1;
    uint zNeighborRight = curZ + 1;

    __syncthreads();
    uint neighborUniqueID;
    if((neighborUniqueID = yDimToFirstNodeIndex[yNeighborLeft + curZ*grid.sizeY]) < numUniqueGridNodes){
        addVoxelDataForUsedNode(neighborUniqueID, sharedBlockStorage,
            nodeIndexToFirstVoxelIndex, voxelIDs, voxelData);
    }
    __syncthreads();
    if((neighborUniqueID = yDimToFirstNodeIndex[yNeighborRight + curZ*grid.sizeY]) < numUniqueGridNodes){
        addVoxelDataForUsedNode(neighborUniqueID, sharedBlockStorage,
            nodeIndexToFirstVoxelIndex, voxelIDs, voxelData);
    }
    __syncthreads();
    if((neighborUniqueID = yDimToFirstNodeIndex[zNeighborRight*grid.sizeY + curY]) < numUniqueGridNodes){
        addVoxelDataForUsedNode(neighborUniqueID, sharedBlockStorage,
            nodeIndexToFirstVoxelIndex, voxelIDs, voxelData);
    }
    __syncthreads();
    if((neighborUniqueID = yDimToFirstNodeIndex[zNeighborLeft*grid.sizeY + curY]) < numUniqueGridNodes){
        addVoxelDataForUsedNode(neighborUniqueID, sharedBlockStorage,
            nodeIndexToFirstVoxelIndex, voxelIDs, voxelData);
    }
    __syncthreads();
}

__global__ void calculateDivU(uint numVoxelsPerNode, uint numVoxels1D, uint numUsedVoxelsInGrid, const uint* nodeIndexToFirstVoxelIndex, const uint* voxelIDs,
                                const float* voxelUs, float radius, uint refinementLevel, Grid grid, const uint* yDimFirstNodeIndex,
                                const uint* gridNodeIndicesToFirstParticleIndex, const uint* gridNodes, float* divU)
{
    extern __shared__ float sharedVoxels[];
    uint index = threadIdx.x + blockIdx.x*blockDim.x;
    if(index < numUsedVoxelsInGrid){
        loadVoxelDataForThisBlock(numVoxelsPerNode, numUsedVoxelsInGrid, sharedVoxels,
                nodeIndexToFirstVoxelIndex, voxelIDs, voxelUs, radius, refinementLevel,
                grid, yDimFirstNodeIndex, gridNodeIndicesToFirstParticleIndex,
                gridNodes);
    }
}

void cudaPressureSolve(const CudaVec<uint>& nodeIndexToFirstVoxelIndex, const CudaVec<uint>& voxelIDs, const CudaVec<float>& voxelUs,
                        float radius, uint refinementLevel, const Grid& grid, const CudaVec<uint>& yDimFirstNodeIndex,
                        const CudaVec<uint>& gridNodeIndicesToFirstParticleIndex, const CudaVec<uint>& gridNodes, uint numVoxelsPerNode,
                        uint numVoxels1D,
                        CudaVec<float> divU, CudaVec<float> Ax, CudaVec<float> Ay, CudaVec<float> Az, CudaVec<float> Adiag, hipStream_t stream)
{
    calculateDivU<<<nodeIndexToFirstVoxelIndex.size(), BLOCKSIZE, sizeof(float)*numVoxelsPerNode, stream>>>
        (numVoxelsPerNode, numVoxels1D, nodeIndexToFirstVoxelIndex.size(), nodeIndexToFirstVoxelIndex.devPtr(),
        voxelIDs.devPtr(), voxelUs.devPtr(), radius, refinementLevel, grid, yDimFirstNodeIndex.devPtr(),
        gridNodeIndicesToFirstParticleIndex.devPtr(), gridNodes.devPtr(), divU.devPtr());
    gpuErrchk(hipPeekAtLastError());
}