#include "hip/hip_runtime.h"
#include "kernels.hu"
#include "parallelPrefixSumKernels.hu"
#include "radixSortKernels.hu"
#include "reductionKernels.hu"
#include "../typedefs.h"


/**
 * @brief find root node containing each particle in domain
 * 
 * @param px 
 * @param py 
 * @param pz 
 * @param numParticles 
 * @param grid 
 * @param gridPosition 
 * @return __global__ 
 */

__global__ void rootCell(float* px, float* py, float* pz, uint numParticles, Grid grid, uint* gridPosition){
    uint index = threadIdx.x + blockIdx.x*blockDim.x;
    if(index < numParticles){
        uint x = floorf((px[index] - grid.negX) / grid.cellSize);
        uint y = floorf((py[index] - grid.negY) / grid.cellSize);
        uint z = floorf((pz[index] - grid.negZ) / grid.cellSize);
        gridPosition[index] = x + y*grid.sizeX + z*grid.sizeX*grid.sizeY;
    }
}

template <typename T>
__device__ T square(T in){
    return in*in;
}

/**
 * @brief Find subcell containing each particle inside its containing grid node
 * 
 * @param px 
 * @param py 
 * @param pz 
 * @param numParticles 
 * @param grid 
 * @param gridPosition 
 * @param subCellPositionX 
 * @param subCellPositionY 
 * @param subCellPositionZ 
 * @param refinementLevel 
 * @param xySize 
 * @return __global__ 
 */

__global__ void subCellCreateNumSubCellsTouchedEachDimension(float* px, float* py, float* pz, uint numParticles, Grid grid, uint* gridPosition, uint* subCellsTouchedX, uint* subCellsTouchedY, uint* subCellsTouchedZ, uint refinementLevel, float radius, uint xySize){
    uint index = threadIdx.x + blockIdx.x*blockDim.x;
    if(index < numParticles){
        uint moduloWRTxySize = gridPosition[index] % xySize;
        uint gridIDz = gridPosition[index] / (xySize);
        uint gridIDy = (moduloWRTxySize) / grid.sizeX;
        uint gridIDx = (moduloWRTxySize) % grid.sizeX;
        
        float pxInGridCell = (px[index] - grid.negX - gridIDx*grid.cellSize);
        float pyInGridCell = (py[index] - grid.negY - gridIDy*grid.cellSize);
        float pzInGridCell = (pz[index] - grid.negZ - gridIDz*grid.cellSize);
        float subCellWidth = grid.cellSize/(2.0f*(1<<refinementLevel));

        uint xTouched = 0;
        uint yTouched = 0;
        uint zTouched = 0;

        uint subCellPositionX = floorf(pxInGridCell/subCellWidth);
        uint subCellPositionY = floorf(pyInGridCell/subCellWidth);
        uint subCellPositionZ = floorf(pzInGridCell/subCellWidth);
        
        uint apronCells = floorf(radius);

        subCellPositionX += apronCells;
        subCellPositionY += apronCells;
        subCellPositionZ += apronCells;

        float halfSubCellWidth = subCellWidth / 2.0f;
        float radiusSCW_squared = square(radius*subCellWidth);

        for(uint x = subCellPositionX - apronCells; x < subCellPositionX + apronCells; ++x){
            for(uint y = subCellPositionY - apronCells; y < subCellPositionY + apronCells; ++y){
                for(uint z = subCellPositionZ - apronCells; z < subCellPositionZ + apronCells; ++z){
                    float subCellBaseX = x * subCellWidth;
                    float subCellBaseY = y * subCellWidth;
                    float subCellBaseZ = z * subCellWidth;

                    if(square(pxInGridCell - subCellBaseX)
                     + square(pyInGridCell - subCellBaseY + halfSubCellWidth)
                     + square(pzInGridCell - subCellBaseZ + halfSubCellWidth) < radiusSCW_squared){
                        ++xTouched;
                    }
                    if(square(pxInGridCell - subCellBaseX + halfSubCellWidth)
                     + square(pyInGridCell - subCellBaseY)
                     + square(pzInGridCell - subCellBaseZ + halfSubCellWidth) < radiusSCW_squared){
                        ++yTouched;
                    }
                    if(square(pxInGridCell - subCellBaseX + halfSubCellWidth)
                     + square(pyInGridCell - subCellBaseY + halfSubCellWidth)
                     + square(pzInGridCell - subCellBaseZ) < radiusSCW_squared){
                        ++zTouched;
                    }
                }
            }
        }

        subCellsTouchedX[index] = xTouched;
        subCellsTouchedY[index] = yTouched;
        subCellsTouchedZ[index] = zTouched;
    }
}

__global__ void subCellCreateLists(float* px, float* py, float* pz, uint numParticles, Grid grid, uint* gridPosition,
        uint* numSubCellsTouchedX, uint* numSubCellsTouchedY, uint* numSubCellsTouchedZ, uint* subCellsX, uint* subCellsY,
        uint* subCellsZ, uint refinementLevel, float radius, uint xySize){
    uint index = threadIdx.x + blockIdx.x*blockDim.x;
    if(index < numParticles){
        uint moduloWRTxySize = gridPosition[index] % xySize;
        uint gridIDz = gridPosition[index] / (xySize);
        uint gridIDy = (moduloWRTxySize) / grid.sizeX;
        uint gridIDx = (moduloWRTxySize) % grid.sizeX;
        
        float pxInGridCell = (px[index] - grid.negX - gridIDx*grid.cellSize);
        float pyInGridCell = (py[index] - grid.negY - gridIDy*grid.cellSize);
        float pzInGridCell = (pz[index] - grid.negZ - gridIDz*grid.cellSize);
        float subCellWidth = grid.cellSize/(2.0f*(1<<refinementLevel));

        uint subCellPositionX = floorf(pxInGridCell/subCellWidth);
        uint subCellPositionY = floorf(pyInGridCell/subCellWidth);
        uint subCellPositionZ = floorf(pzInGridCell/subCellWidth);
        
        uint apronCells = floorf(radius);

        subCellPositionX += apronCells;
        subCellPositionY += apronCells;
        subCellPositionZ += apronCells;

        float halfSubCellWidth = subCellWidth / 2.0f;
        float radiusSCW_squared = square(radius*subCellWidth);

        uint xWritten = 0;
        uint yWritten = 0;
        uint zWritten = 0;

        uint numVoxelsInNodeDimension = 2*apronCells + (2<<refinementLevel);

        uint subCellsTouchedStartX;
        uint subCellsTouchedStartY;
        uint subCellsTouchedStartZ;
        if(index == 0){
            subCellsTouchedStartX = 0;
            subCellsTouchedStartY = 0;
            subCellsTouchedStartZ = 0;
        }
        else{
            subCellsTouchedStartX = numSubCellsTouchedX[index - 1];
            subCellsTouchedStartY = numSubCellsTouchedY[index - 1];
            subCellsTouchedStartZ = numSubCellsTouchedZ[index - 1];
        }

        for(uint x = subCellPositionX - apronCells; x < subCellPositionX + apronCells; ++x){
            for(uint y = subCellPositionY - apronCells; y < subCellPositionY + apronCells; ++y){
                for(uint z = subCellPositionZ - apronCells; z < subCellPositionZ + apronCells; ++z){
                    float subCellBaseX = x * subCellWidth;
                    float subCellBaseY = y * subCellWidth;
                    float subCellBaseZ = z * subCellWidth;

                    if(square(pxInGridCell - subCellBaseX)
                     + square(pyInGridCell - subCellBaseY + halfSubCellWidth)
                     + square(pzInGridCell - subCellBaseZ + halfSubCellWidth) < radiusSCW_squared){
                        subCellsX[subCellsTouchedStartX + xWritten++] = x + y*numVoxelsInNodeDimension + z*numVoxelsInNodeDimension*numVoxelsInNodeDimension;

                    }
                    if(square(pxInGridCell - subCellBaseX + halfSubCellWidth)
                     + square(pyInGridCell - subCellBaseY)
                     + square(pzInGridCell - subCellBaseZ + halfSubCellWidth) < radiusSCW_squared){
                        subCellsY[subCellsTouchedStartY + yWritten++] = y + x*numVoxelsInNodeDimension + z * numVoxelsInNodeDimension * numVoxelsInNodeDimension;
                    }
                    if(square(pxInGridCell - subCellBaseX + halfSubCellWidth)
                     + square(pyInGridCell - subCellBaseY + halfSubCellWidth)
                     + square(pzInGridCell - subCellBaseZ) < radiusSCW_squared){
                        subCellsZ[subCellsTouchedStartZ + zWritten++] = z + x*numVoxelsInNodeDimension + y * numVoxelsInNodeDimension * numVoxelsInNodeDimension;
                    }
                }
            }
        }
    }
}

/**
 * @brief wrapper for rootCell
 * 
 * @param px 
 * @param py 
 * @param pz 
 * @param numParticles 
 * @param grid 
 * @param gridPosition 
 * @param stream 
 */


void kernels::cudaFindGridCell(float* px, float* py, float* pz, uint numParticles, Grid grid, uint* gridPosition, hipStream_t stream){
    rootCell<<<numParticles / BLOCKSIZE + 1, BLOCKSIZE, 0, stream>>>(px, py, pz, numParticles, grid, gridPosition);
}

void kernels::cudaFindSubCell(float* px, float* py, float* pz,
    uint numParticles, Grid grid, uint* gridPosition,
    uint* subCellsTouchedX, uint* subCellsTouchedY,
    uint* subCellsTouchedZ, CudaVec<uint>& subCellPositionX, 
    CudaVec<uint>& subCellPositionY, CudaVec<uint>& subCellPositionZ, 
    uint numRefinementLevels, float radius, hipStream_t stream)
{
    hipStream_t prefixSumStream;
    hipStreamCreate(&prefixSumStream);

    uint* blockSumsSubCells = nullptr;

    hipMallocAsync((void**)&blockSumsSubCells, sizeof(uint)*(numParticles / WORKSIZE + 1), prefixSumStream);

    subCellCreateNumSubCellsTouchedEachDimension<<<numParticles / BLOCKSIZE + 1, BLOCKSIZE, 0, stream>>>
        (px, py, pz, numParticles, grid, gridPosition, subCellsTouchedX, subCellsTouchedY, subCellsTouchedZ,
        numRefinementLevels, radius, grid.sizeX*grid.sizeY);
    

    cudaParallelPrefixSum(numParticles, subCellsTouchedX, blockSumsSubCells, prefixSumStream);
    cudaParallelPrefixSum(numParticles, subCellsTouchedY, blockSumsSubCells, prefixSumStream);
    cudaParallelPrefixSum(numParticles, subCellsTouchedZ, blockSumsSubCells, prefixSumStream);

    hipFreeAsync(blockSumsSubCells, prefixSumStream);
    uint subCellListSizeX[1];
    uint subCellListSizeY[1];
    uint subCellListSizeZ[1];
    hipMemcpyAsync(subCellListSizeX, subCellsTouchedX + numParticles - 1, sizeof(uint), hipMemcpyDeviceToHost, prefixSumStream);
    hipMemcpyAsync(subCellListSizeY, subCellsTouchedY + numParticles - 1, sizeof(uint), hipMemcpyDeviceToHost, prefixSumStream);
    hipMemcpyAsync(subCellListSizeZ, subCellsTouchedZ + numParticles - 1, sizeof(uint), hipMemcpyDeviceToHost, prefixSumStream);
    hipStreamSynchronize(prefixSumStream);

    subCellPositionX.resizeAsync(*subCellListSizeX, stream);
    subCellPositionY.resizeAsync(*subCellListSizeY, stream);
    subCellPositionZ.resizeAsync(*subCellListSizeZ, stream);

    subCellCreateLists<<<numParticles / BLOCKSIZE + 1, BLOCKSIZE, 0, stream>>>(px, py, pz, numParticles, grid, gridPosition,
        subCellsTouchedX, subCellsTouchedY, subCellsTouchedZ, subCellPositionX.devPtr(), subCellPositionY.devPtr(), subCellPositionZ.devPtr(),
        numRefinementLevels, radius, grid.sizeX*grid.sizeY);

    hipStreamDestroy(prefixSumStream);
}

/**
 * @brief Wrapper for executing CUDA parallel prefix sum
 * 
 * @tparam T 
 * @param numElements 
 * @param array 
 * @param blockSums 
 * @param stream 
 */

template <typename T>
void kernels::cudaParallelPrefixSum(uint numElements, T* array, T* blockSums, hipStream_t stream){
    parallelPrefix<<<numElements/WORKSIZE + 1, BLOCKSIZE, 0, stream>>>(numElements, array, blockSums);
    if(numElements / WORKSIZE > 0){
        T* tempBlockSums;
        hipMallocAsync((void**)&tempBlockSums, sizeof(T) * ((numElements/WORKSIZE + 1) / WORKSIZE + 1), stream);
        cudaParallelPrefixSum((numElements/WORKSIZE + 1), blockSums,  tempBlockSums, stream);
        hipFreeAsync(tempBlockSums, stream);
    }
    parallelPrefixApplyPreviousBlockSum<<<numElements/WORKSIZE + 1, WORKSIZE, 0, stream>>>(numElements, array, blockSums);
}

/**
 * @brief Wrapper for performing CUDA radix inclusive sort on uint array
 * 
 * @param numElements 
 * @param inArray 
 * @param outArray 
 * @param sortedIndices 
 * @param front 
 * @param back 
 * @param blockSumsFront 
 * @param blockSumsBack 
 * @param frontStream 
 * @param backStream 
 */

void kernels::cudaRadixSortUint(uint numElements, uint* inArray, uint* outArray, uint* sortedIndices, uint* front, uint* back, hipStream_t frontStream, hipStream_t backStream, uint*& reorderedIndicesRelativeToOriginal){
    uint* tReordered;

    hipMallocAsync((void**)&reorderedIndicesRelativeToOriginal, sizeof(uint) * numElements, backStream); //reordered indices relative to original position, for shuffling positions
    hipMallocAsync((void**)&tReordered, sizeof(uint) * numElements, backStream); //reordered indices relative to original position, for shuffling positions

    for(uint i = 0; i < sizeof(uint)*8; ++i){
        radixBinUintByBitIndex<<<numElements/BLOCKSIZE + 1, BLOCKSIZE, 0, frontStream>>>(numElements, inArray, i, front, back);
        
        uint* blockSumsFront;
        uint* blockSumsBack;
        hipMallocAsync((void**)&blockSumsFront, sizeof(uint)*numElements/WORKSIZE + 1, frontStream);
        hipMallocAsync((void**)&blockSumsBack, sizeof(uint)*numElements/WORKSIZE + 1, backStream);

        hipStreamSynchronize(frontStream);
        kernels::cudaParallelPrefixSum<uint>(numElements, front, blockSumsFront, frontStream);
        kernels::cudaParallelPrefixSum<uint>(numElements, back, blockSumsBack, backStream);

        hipFreeAsync(blockSumsFront, frontStream);
        hipFreeAsync(blockSumsBack, backStream);

        hipStreamSynchronize(backStream);
        coalesceFrontBack<<<numElements/BLOCKSIZE + 1, BLOCKSIZE, 0, frontStream>>>(numElements, sortedIndices, front, back);
        reorderGridIndices<<<numElements/BLOCKSIZE + 1, BLOCKSIZE, 0, frontStream>>>(numElements, sortedIndices, inArray, outArray);
        hipStreamSynchronize(frontStream);
        
        if(i == 0){
            hipMemcpyAsync(reorderedIndicesRelativeToOriginal, sortedIndices, sizeof(uint)*numElements, hipMemcpyDeviceToDevice, frontStream);
        }
        else{
            reorderGridIndices<<<numElements/BLOCKSIZE + 1, BLOCKSIZE, 0, frontStream>>>(numElements, sortedIndices, reorderedIndicesRelativeToOriginal, tReordered);
            uint* temp = tReordered;
            tReordered = reorderedIndicesRelativeToOriginal;
            reorderedIndicesRelativeToOriginal = temp;
        }

        uint* tempGP = inArray;
        inArray = outArray;
        outArray = tempGP;
    }
    hipFreeAsync(tReordered, backStream);
}

/**
 * @brief Sort particles globally by containing root nodes
 * 
 * @param numParticles 
 * @param gridPosition 
 * @param stream 
 */

void kernels::cudaSortParticlesByGridNode(uint numParticles, uint*& gridPosition, uint*& reorderedIndicesRelativeToOriginal, hipStream_t stream){
    uint* ogGridPosition = gridPosition;
    uint* ogReordered = reorderedIndicesRelativeToOriginal;

    uint* sortedGridPosition;
    uint* sortedParticleIndices;
    uint* front;
    uint* back;

    hipMallocAsync((void**)&sortedGridPosition, sizeof(uint)*numParticles, stream);
    hipMallocAsync((void**)&sortedParticleIndices, sizeof(uint)*numParticles, stream);
    hipMallocAsync((void**)&front, sizeof(uint)*numParticles, stream);
    hipMallocAsync((void**)&back, sizeof(uint)*numParticles, stream);

    hipStream_t backStream;
    hipStreamCreate(&backStream);
    
    cudaRadixSortUint(numParticles, gridPosition, sortedGridPosition, sortedParticleIndices, front, back, stream, backStream, reorderedIndicesRelativeToOriginal);

    if(ogGridPosition != sortedGridPosition){
        hipFreeAsync(sortedGridPosition, stream);
    }

    // if(ogReordered != reorderedIndicesRelativeToOriginal){
        // hipFree(reorderedIndicesRelativeToOriginal);
    // }

    hipFreeAsync(sortedParticleIndices, stream);
    hipFreeAsync(front, stream);
    hipFreeAsync(back, stream);
}

__global__ void markUniqueGridCells(uint numElements, uint* gridCells, uint* uniqueGridNodes){
    uint index = threadIdx.x + blockDim.x * blockIdx.x;
    if(index < numElements){
        if(index == 0){
            uniqueGridNodes[index] = 1;
        }
        else if(gridCells[index] != gridCells[index - 1]){
            uniqueGridNodes[index] = 1;
        }
        else{
            uniqueGridNodes[index] = 0;
        }
    }
}

uint kernels::cudaMarkUniqueGridCellsAndCount(uint numParticles, uint* gridCells, uint* uniqueGridNodes, hipStream_t stream){
    markUniqueGridCells<<<numParticles / BLOCKSIZE + 1, BLOCKSIZE, 0, stream>>>(numParticles, gridCells, uniqueGridNodes);
    
    uint* uniqueBlockSums;
    uint numGridNodes[1];
    
    hipMallocAsync((void**)&uniqueBlockSums, sizeof(uint) * (numParticles / WORKSIZE + 1), stream);
    cudaParallelPrefixSum(numParticles,uniqueGridNodes, uniqueBlockSums, stream);
    hipMemcpyAsync(numGridNodes, uniqueGridNodes + numParticles - 1, sizeof(uint), hipMemcpyDeviceToHost, stream);

    hipFreeAsync(uniqueBlockSums, stream);

    return numGridNodes[0];
}

__global__ void mapNodeIndicesToParticles(uint numParticles, uint* uniqueGridNodes, uint* gridNodeIndicesToFirstParticleIndex){
    uint index = threadIdx.x + blockDim.x*blockIdx.x;
    if(index < numParticles){
        if(index == 0){
            gridNodeIndicesToFirstParticleIndex[0] = 0;
        }
        else{
            if(uniqueGridNodes[index] != uniqueGridNodes[index - 1]){
                gridNodeIndicesToFirstParticleIndex[uniqueGridNodes[index] - 1] = index;
            }
        }
    }
}

void kernels::cudaMapNodeIndicesToParticles(uint numParticles, uint* uniqueGridNodes, uint* gridNodeIndicesToFirstParticleIndex, hipStream_t stream){
    mapNodeIndicesToParticles<<<numParticles / BLOCKSIZE + 1, BLOCKSIZE, 0, stream>>>(numParticles, uniqueGridNodes, gridNodeIndicesToFirstParticleIndex);
}

__global__ void sumParticlesPerNode(uint numGridNodes, uint numParticles, uint* gridNodeIndicesToFirstParticleIndex, uint* subCellsTouchedPerParticle,
        uint* subCellsDim, uint* numNonZeroVoxels, uint* numParticlesInVoxelLists, uint numVoxelsPerNode){
    uint index = threadIdx.x + blockDim.x*blockIdx.x;

    extern __shared__ uint voxelCount[];
    __shared__ uint sums[BLOCKSIZE];
    __shared__ uint maxParticleNum;

    if(threadIdx.x == 0){
        if(blockIdx.x < numGridNodes - 1){
            maxParticleNum = gridNodeIndicesToFirstParticleIndex[blockIdx.x + 1];
        }
        else{
            maxParticleNum = numParticles;
        }
    }
    for(uint i = threadIdx.x; i < numVoxelsPerNode; i += blockDim.x){
        voxelCount[i] = 0;
    }

    __syncthreads();

    for(uint i = gridNodeIndicesToFirstParticleIndex[blockIdx.x] + threadIdx.x; i < maxParticleNum; i += blockDim.x){
        for(uint j = subCellsTouchedPerParticle[i]; j < subCellsTouchedPerParticle[i+1]; ++j){
            atomicAdd(voxelCount + subCellsDim[j], 1);
        }
    }
}

void kernels::cudaSumParticlesPerNodeAndWriteNumUsedVoxels(uint numGridNodes, uint numParticles, uint* gridNodeIndicesToFirstParticleIndex, uint* subCellsTouchedPerParticle, uint* subCellsDim, uint* numNonZeroVoxels,
    uint* numParticlesInVoxelLists, uint numVoxelsPerNode, hipStream_t stream){
    sumParticlesPerNode<<<numGridNodes, BLOCKSIZE, sizeof(uint) * numVoxelsPerNode, stream>>>(
        numGridNodes, numParticles, gridNodeIndicesToFirstParticleIndex, subCellsTouchedPerParticle, subCellsDim, numNonZeroVoxels,
        numParticlesInVoxelLists, numVoxelsPerNode);
    gpuErrchk(hipPeekAtLastError());
        
    }