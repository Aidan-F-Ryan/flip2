#include "hip/hip_runtime.h"
//Copyright 2023 Aberrant Behavior LLC

#include "cudaVec.hu"
#include "typedefs.h"
#include <cstdlib>
#include <iostream>
#include "cudaDeviceManager.hu"

template <typename T>
CudaVec<T>::CudaVec()
: d_vec(nullptr){

}

template <typename T>
CudaVec<T>::CudaVec(const uint& size){
    resize(size);
}

template <typename T>
void CudaVec<T>::resize(const uint& size){
    if(d_vec != nullptr){
        GPU_MEMORY_ALLOCATED -= sizeof(T) * vec.size();
    }
    numElements = size;
    vec.resize(size);
    cuMalloc();
}

template <typename T>
void CudaVec<T>::resizeAsync(const uint& size, const hipStream_t& stream){
    if(d_vec != nullptr){
        GPU_MEMORY_ALLOCATED -= sizeof(T) * vec.size();
    }
    numElements = size;
    cuMallocAsync(stream);
}

template <typename T>
T& CudaVec<T>::operator[](const uint& i){
    return vec[i];
}

template <typename T>
const T& CudaVec<T>::operator[](const uint& i) const{
    return vec[i];
}

template <typename T>
void CudaVec<T>::upload(){
    gpuErrchk( hipMemcpy(d_vec, vec.data(), vec.size()*sizeof(T), hipMemcpyHostToDevice) );
}

template <typename T>
void CudaVec<T>::download(){
    if(numElements != vec.size()){
        vec.resize(numElements);
    }
    gpuErrchk( hipMemcpy(vec.data(), d_vec, vec.size()*sizeof(T), hipMemcpyDeviceToHost) );
}

template <typename T>
void CudaVec<T>::upload(hipStream_t stream){
    if(numElements != vec.size()){
        std::cerr<<"ERROR: ATTEMPT TO UPLOAD CPU VECTOR TO ASYNC ALLOCATED GPU VECTOR\nCPU VEC SIZE: "<<vec.size()<<"\nGPU_VEC SIZE: "<<numElements<<"\n";
        exit(1);
    }
    gpuErrchk( hipMemcpyAsync(d_vec, vec.data(), vec.size()*sizeof(T), hipMemcpyHostToDevice, stream) );
}

template <typename T>
void CudaVec<T>::download(hipStream_t stream){
    if(vec.size() != numElements){
        vec.resize(numElements);
    }
    gpuErrchk( hipMemcpyAsync(vec.data(), d_vec, vec.size()*sizeof(T), hipMemcpyDeviceToHost, stream) );
}

template <typename T>
T* CudaVec<T>::devPtr(){
    return d_vec;
}

template <typename T>
const T* CudaVec<T>::devPtr() const{
    return d_vec;
}

template<typename T>
void CudaVec<T>::cuMalloc(){
    if(d_vec != nullptr){
        gpuErrchk( hipFree(d_vec) );
    }
    gpuErrchk( hipMalloc((void**)&d_vec, sizeof(T)*vec.size()) );
    GPU_MEMORY_ALLOCATED += sizeof(T) * vec.size();
}

template<typename T>
void CudaVec<T>::cuMallocAsync(const hipStream_t& stream){
    if(d_vec != nullptr){
        gpuErrchk( hipFreeAsync(d_vec, stream) );
    }
    gpuErrchk( hipMallocAsync((void**)&d_vec, sizeof(T)*numElements, stream) );
    GPU_MEMORY_ALLOCATED += sizeof(T) * numElements;
}

template<typename T>
void CudaVec<T>::swapDevicePtr(T* devPtr){
    if(d_vec != nullptr && d_vec != devPtr){
        gpuErrchk( hipFree(d_vec) );
        GPU_MEMORY_ALLOCATED -= sizeof(T) * numElements;
    }
    d_vec = devPtr;
}

template<typename T>
void CudaVec<T>::swapDevicePtrAsync(T* devPtr, hipStream_t stream){
    if(d_vec != nullptr && d_vec != devPtr){
        gpuErrchk( hipFreeAsync(d_vec, stream) );
        GPU_MEMORY_ALLOCATED -= sizeof(T) * numElements;
    }
    d_vec = devPtr;
}

template <typename T>
void CudaVec<T>::clear(){
    if(d_vec != nullptr){
        GPU_MEMORY_ALLOCATED -= sizeof(T) * numElements;
        hipFree(d_vec);
        d_vec = nullptr;
    }
    vec.clear();
    numElements = 0;
}

template <typename T>
void CudaVec<T>::clearAsync(hipStream_t stream){
    if(d_vec != nullptr){
        GPU_MEMORY_ALLOCATED -= sizeof(T) * numElements;
        hipFreeAsync(d_vec, stream);
        d_vec = nullptr;
    }
    vec.clear();
    numElements = 0;
}

template<typename T>
void CudaVec<T>::print() const{
    for(uint i = 0; i < vec.size(); ++i){
        std::cout<<+vec[i]<<", ";
    }
    std::cout<<std::endl;
}

template<typename T>
void CudaVec<T>::zeroDevice(){
    zeroArray<<<numElements / BLOCKSIZE + 1, BLOCKSIZE>>>(numElements, d_vec);
}

template<typename T>
void CudaVec<T>::zeroDeviceAsync(hipStream_t stream){
    zeroArray<<<numElements / BLOCKSIZE + 1, BLOCKSIZE, 0, stream>>>(numElements, d_vec);
}

__device__ char abs(const char& in){
    return in;
}

__device__ uint abs(const uint& in){
    return in;
}

template<typename T>
__global__ void getMaxFromArray(bool absolute, uint numElements, T* array, T* out){
    __shared__ T shared[WORKSIZE];
    uint index = threadIdx.x + blockIdx.x*WORKSIZE;
    if(index < numElements){
        shared[threadIdx.x] = array[index];
    }
    else{
        shared[threadIdx.x] = 0;
    }
    if(index + blockDim.x < numElements){
        shared[threadIdx.x + blockDim.x] = array[index + blockDim.x];
    }
    else{
        shared[threadIdx.x + blockDim.x] = 0;
    }

    for(int i = blockDim.x; i >= 1; i >>= 1){
        __syncthreads();
        if(!absolute)
            shared[threadIdx.x] = shared[threadIdx.x] > shared[threadIdx.x + blockDim.x] ? shared[threadIdx.x] : shared[threadIdx.x + blockDim.x];
        else
            shared[threadIdx.x] = abs(shared[threadIdx.x]) > abs(shared[threadIdx.x + blockDim.x]) ? shared[threadIdx.x] : shared[threadIdx.x + blockDim.x];
    }

    __syncthreads();

    if(threadIdx.x == 0){
        out[blockIdx.x] = shared[threadIdx.x];
    }
}

template <typename T>
T CudaVec<T>::getMax(hipStream_t stream, bool abs){
    T* outArray;
    T* outArray2;
    T out;
    hipMallocAsync((void**)&outArray, sizeof(T) * (numElements / WORKSIZE + 1), stream);
    gpuErrchk(hipPeekAtLastError());
    getMaxFromArray<<<numElements / WORKSIZE + 1, BLOCKSIZE, 0, stream>>>(abs, numElements, d_vec, outArray);
    gpuErrchk(hipPeekAtLastError());
    hipMallocAsync((void**)&outArray2, sizeof(T) * ((numElements / WORKSIZE + 1) / WORKSIZE + 1), stream);
    gpuErrchk(hipPeekAtLastError());
    hipStreamSynchronize(stream);
    for(int i = numElements / WORKSIZE + 1; i > 1; i = i / WORKSIZE + 1){
        getMaxFromArray<<<i / WORKSIZE + 1, BLOCKSIZE, 0, stream>>>(abs, i, outArray, outArray2);
        gpuErrchk(hipPeekAtLastError());
        hipStreamSynchronize(stream);
        T* temp = outArray;
        outArray = outArray2;
        outArray2 = temp;
    }
    hipMemcpyAsync(&out, outArray, sizeof(T), hipMemcpyDeviceToHost, stream);
    gpuErrchk(hipPeekAtLastError());
    hipStreamSynchronize(stream);
    hipFreeAsync(outArray, stream);
    hipFreeAsync(outArray2, stream);
    hipStreamSynchronize(stream);
    gpuErrchk(hipPeekAtLastError());
    return out;
}

template<typename T>
CudaVec<T>::~CudaVec(){
    if(d_vec != nullptr){
        GPU_MEMORY_ALLOCATED -= sizeof(T) * vec.size();
        gpuErrchk( hipFree(d_vec) );
    }
}

template class CudaVec<float>;
template class CudaVec<uint>;
template class CudaVec<char>;
// template class CudaVec<bool>;

template <typename T>
uint CudaVec<T>::GPU_MEMORY_ALLOCATED = 0;