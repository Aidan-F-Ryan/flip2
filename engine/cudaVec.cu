#include "cudaVec.hu"
#include <iostream>

template <typename T>
CudaVec<T>::CudaVec()
: d_vec(0){

}

template <typename T>
CudaVec<T>::CudaVec(const uint& size){
    resize(size);
}

template <typename T>
void CudaVec<T>::resize(const uint& size){
    numElements = size;
    vec.resize(size);
    cuMalloc();
}

template <typename T>
void CudaVec<T>::resizeAsync(const uint& size, const hipStream_t& stream){
    numElements = size;
    cuMallocAsync(stream);
}

template <typename T>
T& CudaVec<T>::operator[](const uint& i){
    return vec[i];
}

template <typename T>
const T& CudaVec<T>::operator[](const uint& i) const{
    return vec[i];
}

template <typename T>
void CudaVec<T>::upload(){
    hipMemcpy(d_vec, vec.data(), vec.size()*sizeof(T), hipMemcpyHostToDevice);
}

template <typename T>
void CudaVec<T>::download(){
    hipMemcpy(vec.data(), d_vec, vec.size()*sizeof(T), hipMemcpyDeviceToHost);
}

template <typename T>
void CudaVec<T>::upload(hipStream_t stream){
    hipMemcpyAsync(d_vec, vec.data(), vec.size()*sizeof(T), hipMemcpyHostToDevice, stream);
}

template <typename T>
void CudaVec<T>::download(hipStream_t stream){
    if(vec.size() != numElements){
        vec.resize(numElements);
    }
    hipMemcpyAsync(vec.data(), d_vec, vec.size()*sizeof(T), hipMemcpyDeviceToHost, stream);
}

template <typename T>
T* CudaVec<T>::devPtr(){
    return d_vec;
}

template<typename T>
void CudaVec<T>::cuMalloc(){
    if(d_vec != nullptr){
        hipFree(d_vec);
    }
    hipMalloc((void**)&d_vec, sizeof(T)*vec.size());
}

template<typename T>
void CudaVec<T>::cuMallocAsync(const hipStream_t& stream){
    if(d_vec != nullptr){
        hipFreeAsync(d_vec, stream);
    }
    hipMallocAsync((void**)&d_vec, sizeof(T)*numElements, stream);
}

template<typename T>
void CudaVec<T>::swapDevicePtr(T* devPtr){
    if(d_vec != nullptr && d_vec != devPtr){
        hipFree(d_vec);
    }
    d_vec = devPtr;
}

template<typename T>
void CudaVec<T>::print() const{
    for(uint i = 0; i < vec.size(); ++i){
        std::cout<<+vec[i]<<", ";
    }
    std::cout<<std::endl;
}

template<typename T>
CudaVec<T>::~CudaVec(){
    if(d_vec != nullptr){
        hipFree(d_vec);
    }
}

template class CudaVec<float>;
template class CudaVec<uint>;
template class CudaVec<char>;