#include "hip/hip_runtime.h"
#include "kernels.hu"
#include "parallelPrefixSumKernels.hu"
#include "radixSortKernels.hu"

#include <stdio.h>

__global__ void rootCell(float* px, float* py, float* pz, uint numParticles, Grid grid, uint* gridPosition){
    uint index = threadIdx.x + blockIdx.x*blockDim.x;
    if(index < numParticles){
        uint x = floorf((px[index] - grid.negX) / grid.cellSize);
        uint y = floorf((py[index] - grid.negY) / grid.cellSize);
        uint z = floorf((pz[index] - grid.negZ) / grid.cellSize);
        gridPosition[index] = x + y*grid.sizeX + z*grid.sizeX*grid.sizeY;
    }
}

__global__ void subCell(float* px, float* py, float* pz, uint numParticles, Grid grid, uint* gridPosition, uint* subCellPositionX, uint* subCellPositionY, uint* subCellPositionZ, uint refinementLevel, uint xySize){
    uint index = threadIdx.x + blockIdx.x*blockDim.x;
    if(index < numParticles){
        uint moduloWRTxySize = gridPosition[index] % xySize;
        uint z = gridPosition[index] / (xySize);
        uint y = (moduloWRTxySize) / grid.sizeX;
        uint x = (moduloWRTxySize) % grid.sizeX;
        
        float gridCellPositionX = (px[index] - grid.negX - x*grid.cellSize);
        float gridCellPositionY = (py[index] - grid.negY - y*grid.cellSize);
        float gridCellPositionZ = (pz[index] - grid.negZ - z*grid.cellSize);
        float curSubCellSize = grid.cellSize/(2.0f*(1<<refinementLevel));

        subCellPositionX[index] = floorf(gridCellPositionX/curSubCellSize);
        subCellPositionY[index] = floorf(gridCellPositionY/curSubCellSize);
        subCellPositionZ[index] = floorf(gridCellPositionZ/curSubCellSize);
        
    }
}


void kernels::cudaFindGridCell(float* px, float* py, float* pz, uint numParticles, Grid grid, uint* gridPosition, hipStream_t stream){
    rootCell<<<numParticles / BLOCKSIZE + 1, BLOCKSIZE, 0, stream>>>(px, py, pz, numParticles, grid, gridPosition);
}

void kernels::cudaFindSubCell(float* px, float* py, float* pz, uint numParticles, Grid grid, uint* gridPosition, uint* subCellPositionX, uint* subCellPositionY, uint* subCellPositionZ, uint numRefinementLevels, hipStream_t stream){
    subCell<<<numParticles / BLOCKSIZE + 1, BLOCKSIZE, 0, stream>>>(px, py, pz, numParticles, grid, gridPosition, subCellPositionX, subCellPositionY, subCellPositionZ, numRefinementLevels, grid.sizeX*grid.sizeY);
}

template <typename T>
void kernels::cudaParallelPrefixSum(uint numElements, T* array, T* blockSums, hipStream_t stream){
    parallelPrefix<<<numElements/BLOCKSIZE + 1, BLOCKSIZE, 0, stream>>>(numElements, array, blockSums);
    parallelPrefixApplyPreviousBlockSum<<<numElements/BLOCKSIZE + 1, BLOCKSIZE, 0, stream>>>(numElements, array, blockSums);
}

void kernels::cudaRadixSortUint(uint numElements, uint* inArray, uint* outArray, uint* sortedIndices, uint* front, uint* back, uint* blockSumsFront, uint* blockSumsBack, hipStream_t frontStream, hipStream_t backStream){
    for(uint i = 0; i < sizeof(uint)*8; ++i){
        radixBinUintByBitIndex<<<numElements/BLOCKSIZE + 1, BLOCKSIZE, 0, frontStream>>>(numElements, inArray, i, front, back);
        hipStreamSynchronize(frontStream);
        kernels::cudaParallelPrefixSum<uint>(numElements, front, blockSumsFront, frontStream);
        kernels::cudaParallelPrefixSum<uint>(numElements, back, blockSumsBack, backStream);
        hipStreamSynchronize(backStream);
        coalesceFrontBack<<<numElements/BLOCKSIZE + 1, BLOCKSIZE, 0, frontStream>>>(numElements, sortedIndices, front, back);
        reorderGridIndices<<<numElements/BLOCKSIZE + 1, BLOCKSIZE, 0, frontStream>>>(numElements, sortedIndices, inArray, outArray);
        hipStreamSynchronize(frontStream);
        uint* tempGP = inArray;
        inArray = outArray;
        outArray = tempGP;
    }
}

void kernels::cudaSortParticlesByGridNode(uint numParticles, uint*& gridPosition, hipStream_t stream){
    uint* ogGridPosition = gridPosition;

    uint* sortedGridPosition;
    uint* sortedParticleIndices;
    uint* front;
    uint* back;
    uint* blockSumsFront;
    uint* blockSumsBack;

    hipMalloc((void**)&sortedGridPosition, sizeof(uint)*numParticles);
    hipMalloc((void**)&sortedParticleIndices, sizeof(uint)*numParticles);
    hipMalloc((void**)&front, sizeof(uint)*numParticles);
    hipMalloc((void**)&back, sizeof(uint)*numParticles);
    hipMalloc((void**)&blockSumsFront, sizeof(uint)*numParticles/BLOCKSIZE + 1);
    hipMalloc((void**)&blockSumsBack, sizeof(uint)*numParticles/BLOCKSIZE + 1);

    hipStream_t backStream;
    hipStreamCreate(&backStream);
    
    cudaRadixSortUint(numParticles, gridPosition, sortedGridPosition, sortedParticleIndices, front, back, blockSumsFront, blockSumsBack, stream, backStream);
    
    if(ogGridPosition != sortedGridPosition){
        hipFree(sortedGridPosition);
    }

    hipFree(sortedParticleIndices);
    hipFree(front);
    hipFree(back);
    hipFree(blockSumsFront);
    hipFree(blockSumsBack);
}