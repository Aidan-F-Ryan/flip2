#include "hip/hip_runtime.h"
#include "kernels.hu"
#include "parallelPrefixSumKernels.hu"

#include <stdio.h>

__global__ void rootCell(float* px, float* py, float* pz, uint numParticles, Grid grid, uint* gridPosition){
    uint index = threadIdx.x + blockIdx.x*blockDim.x;
    if(index < numParticles){
        uint x = floorf((px[index] - grid.negX) / grid.cellSize);
        uint y = floorf((py[index] - grid.negY) / grid.cellSize);
        uint z = floorf((pz[index] - grid.negZ) / grid.cellSize);
        gridPosition[index] = x + y*grid.sizeX + z*grid.sizeX*grid.sizeY;
    }
}

__global__ void subCell(float* px, float* py, float* pz, uint numParticles, Grid grid, uint* gridPosition, uint* subCellPositionX, uint* subCellPositionY, uint* subCellPositionZ, uint refinementLevel, uint xySize){
    uint index = threadIdx.x + blockIdx.x*blockDim.x;
    if(index < numParticles){
        uint moduloWRTxySize = gridPosition[index] % xySize;
        uint z = gridPosition[index] / (xySize);
        uint y = (moduloWRTxySize) / grid.sizeX;
        uint x = (moduloWRTxySize) % grid.sizeX;
        
        float gridCellPositionX = (px[index] - grid.negX - x*grid.cellSize);
        float gridCellPositionY = (py[index] - grid.negY - y*grid.cellSize);
        float gridCellPositionZ = (pz[index] - grid.negZ - z*grid.cellSize);
        float curSubCellSize = grid.cellSize/(2.0f*(1<<refinementLevel));

        subCellPositionX[index] = floorf(gridCellPositionX/curSubCellSize);
        subCellPositionY[index] = floorf(gridCellPositionY/curSubCellSize);
        subCellPositionZ[index] = floorf(gridCellPositionZ/curSubCellSize);
        
    }
}

__global__ void radixBinParticlesByGridPositionBitIndex(uint numParticles, uint* gridPosition, uint bitIndex, uint* front, uint* back){
    uint index = threadIdx.x + blockIdx.x*blockDim.x;
    if(index < numParticles){
        if((gridPosition[index] & (1<<bitIndex)) == 0){
            front[index] = 1;
            back[index] = 0;
        }
        else{
            front[index] = 0;
            back[index] = 1;
        }
    }
    
}


__global__ void coalesceFrontBack(uint numParticles, uint* sortedParticleIndices, uint* front, uint* back){
    uint index = threadIdx.x + blockIdx.x*blockDim.x;
    __shared__ uint maxFrontIndex;
    __shared__ uint thisBlockFront[BLOCKSIZE+1];
    __shared__ uint thisBlockBack[BLOCKSIZE+1];
    if(threadIdx.x == 0 && index < numParticles){
        maxFrontIndex = front[numParticles-1];
        if((int)index - 1 >= 0){
            thisBlockFront[0] = front[index - 1];
            thisBlockBack[0] = back[index - 1];
        }
        else{
            thisBlockFront[0] = 0;
            thisBlockBack[0] = 0;
        }
    }
    __syncthreads();
    if(index < numParticles){
        thisBlockFront[threadIdx.x + 1] = front[index];
        thisBlockBack[threadIdx.x + 1] = back[index];
    }
    __syncthreads();
    if(index < numParticles){
        if(thisBlockFront[threadIdx.x] != thisBlockFront[threadIdx.x+1]){
            sortedParticleIndices[thisBlockFront[threadIdx.x]] = index;
        }
        if(thisBlockBack[threadIdx.x] != thisBlockBack[threadIdx.x+1]){
            sortedParticleIndices[thisBlockBack[threadIdx.x] + maxFrontIndex] = index;
        }
    }
}

__global__ void reorderGridIndices(uint numParticles, uint* sortedParticleIndices, uint* gridPosition, uint* sortedGridPosition){
    uint index = threadIdx.x + blockIdx.x*blockDim.x;
    if(index < numParticles){
        sortedGridPosition[index] = gridPosition[sortedParticleIndices[index]];
    }
}

void kernels::cudaFindGridCell(float* px, float* py, float* pz, uint numParticles, Grid grid, uint* gridPosition, hipStream_t stream){
    rootCell<<<numParticles / BLOCKSIZE + 1, BLOCKSIZE, 0, stream>>>(px, py, pz, numParticles, grid, gridPosition);
}

void kernels::cudaFindSubCell(float* px, float* py, float* pz, uint numParticles, Grid grid, uint* gridPosition, uint* subCellPositionX, uint* subCellPositionY, uint* subCellPositionZ, uint numRefinementLevels, hipStream_t stream){
    subCell<<<numParticles / BLOCKSIZE + 1, BLOCKSIZE, 0, stream>>>(px, py, pz, numParticles, grid, gridPosition, subCellPositionX, subCellPositionY, subCellPositionZ, numRefinementLevels, grid.sizeX*grid.sizeY);
}

template <typename T>
void kernels::cudaParallelPrefixSum(uint numElements, T* array, T* blockSums, hipStream_t stream){
    parallelPrefix<<<numElements/BLOCKSIZE + 1, BLOCKSIZE, 0, stream>>>(numElements, array, blockSums);
    parallelPrefixApplyPreviousBlockSum<<<numElements/BLOCKSIZE + 1, BLOCKSIZE, 0, stream>>>(numElements, array, blockSums);
}

void kernels::cudaSortParticles(uint numParticles, uint*& gridPosition, hipStream_t stream){
    uint* ogGridPosition = gridPosition;

    uint* sortedGridPosition;
    uint* sortedParticleIndices;
    uint* front;
    uint* back;
    uint* blockSumsFront;
    uint* blockSumsBack;

    hipMalloc((void**)&sortedGridPosition, sizeof(uint)*numParticles);
    hipMalloc((void**)&sortedParticleIndices, sizeof(uint)*numParticles);
    hipMalloc((void**)&front, sizeof(uint)*numParticles);
    hipMalloc((void**)&back, sizeof(uint)*numParticles);
    hipMalloc((void**)&blockSumsFront, sizeof(uint)*numParticles/BLOCKSIZE + 1);
    hipMalloc((void**)&blockSumsBack, sizeof(uint)*numParticles/BLOCKSIZE + 1);

    hipStream_t backStream;
    hipStreamCreate(&backStream);
    for(uint i = 0; i < sizeof(uint)*8; ++i){
        radixBinParticlesByGridPositionBitIndex<<<numParticles/BLOCKSIZE + 1, BLOCKSIZE, 0, stream>>>(numParticles, gridPosition, i, front, back);
        hipStreamSynchronize(stream);
        kernels::cudaParallelPrefixSum<uint>(numParticles, front, blockSumsFront, stream);
        kernels::cudaParallelPrefixSum<uint>(numParticles, back, blockSumsBack, backStream);
        hipStreamSynchronize(backStream);
        coalesceFrontBack<<<numParticles/BLOCKSIZE + 1, BLOCKSIZE, 0, stream>>>(numParticles, sortedParticleIndices, front, back);
        reorderGridIndices<<<numParticles/BLOCKSIZE + 1, BLOCKSIZE, 0, stream>>>(numParticles, sortedParticleIndices, gridPosition, sortedGridPosition);
        hipStreamSynchronize(stream);
        uint* tempGP = gridPosition;
        gridPosition = sortedGridPosition;
        sortedGridPosition = tempGP;
    }
    if(ogGridPosition != sortedGridPosition){
        hipFree(sortedGridPosition);
    }

    hipFree(sortedParticleIndices);
    hipFree(front);
    hipFree(back);
    hipFree(blockSumsFront);
    hipFree(blockSumsBack);
}